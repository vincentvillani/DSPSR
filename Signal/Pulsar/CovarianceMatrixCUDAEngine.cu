#include "hip/hip_runtime.h"
/*
 * CovarianceMatrixEngineCUDA.C
 *
 *  Created on: 01/09/2014
 *      Author: vincentvillani
 */

#include "dsp/CovarianceMatrixCUDAEngine.h"

//TODO: VINCENT: ADD A HITS CHAN == 1 VARIATION TO STOP NEEDLESS COPYIES


CovarianceMatrixCUDAEngine::CovarianceMatrixCUDAEngine()
{
	h_zeroes = new bool;
	hipMalloc(&d_zeroes, sizeof(bool));
}


CovarianceMatrixCUDAEngine::~CovarianceMatrixCUDAEngine()
{
	delete h_zeroes;
	hipFree(d_zeroes);
}



void CovarianceMatrixCUDAEngine::computeCovarianceMatrixCUDAEngine(float* d_result, unsigned int resultElementOffset,
	const float* h_amps, float* d_amps, unsigned int ampsLength,
	const unsigned int* h_hits, unsigned int* d_hits, unsigned int hitsLength,
	unsigned int stokesLength, unsigned int blockDim2D)
{

	hipMemcpy(d_hits, h_hits, sizeof(unsigned int) * hitsLength, hipMemcpyHostToDevice);

	//If there are bins with zeroes, discard everything
	if ( hitsContainsZeroes(d_hits, hitsLength) )
		return;


	//printf("RUNNING KERNELS\n");

	int meanBlockDim = blockDim2D * blockDim2D;
	int meanGridDim = ceil((float) ampsLength / meanBlockDim);

	//Copy new amps and hit data to the device
	hipMemcpy(d_amps, h_amps, sizeof(float) * ampsLength, hipMemcpyHostToDevice);


	printf("Launching Mean Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);
	meanStokesKernel<<< meanGridDim, meanBlockDim >>>(d_amps, ampsLength, d_hits, stokesLength);

	//TODO: DEBUG
	hipError_t error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(1);
	}

	//Compute the needed block and grid dimensions
	int blockDimX = blockDim2D;
	int blockDimY = blockDim2D;
	int gridDimX = ceil((float) ampsLength / blockDimX);
	int gridDimY = ceil((float) ((ampsLength / 2) + 1) / blockDimY);

	dim3 grid = dim3(gridDimX, gridDimY);
	dim3 block = dim3(blockDimX, blockDimY);

	//Call the kernel
	//Compute covariance matrix
	printf("Launching outerProduct Kernel with gridDim: (%d, %d), blockDim: (%d, %d)\n\n",
			grid.x, grid.y, block.x, block.y);
	outerProductKernel<<< grid, block >>>(d_result + resultElementOffset, d_amps, ampsLength);

	//TODO: DEBUG
	error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(2);
	}

}



void CovarianceMatrixCUDAEngine::compute_final_covariance_matrices_device(
		float* d_outerProducts, unsigned int outerProductsLength,
		float* d_runningMeanSum, unsigned int runningMeanSumLength,
		unsigned int unloadCalledCount, unsigned int freqChanNum,
		unsigned int covarianceLength, unsigned int ampsLength)
{
	//check available memory
	size_t freeMemoryBytes;
	size_t totalMemoryBytes;

	hipMemGetInfo(&freeMemoryBytes, &totalMemoryBytes);

	printf("Free memory: %d\nTotal Memory: %d\n", freeMemoryBytes, totalMemoryBytes);
}




float* CovarianceMatrixCUDAEngine::compute_outer_product_phase_series_device(float* d_runningMeanSum, unsigned int runningMeanSumLength,
			unsigned int unloadCalledCount, unsigned int freqChanNum, unsigned int covarianceLength, unsigned int ampsLength)
{
	/*
	float* d_outerProduct;
	hipMalloc(&d_outerProduct, sizeof(float) * freqChanNum * covarianceLength);

	//divide the running mean by the number of times unload was called
	unsigned int blockDim = 256;
	unsigned int gridDim = ceil(runningMeanSumLength / blockDim);

	genericDivideKernel<<< gridDim, blockDim >>> (runningMeanSumLength, d_runningMeanSum, unloadCalledCount);

	//Do the outer product

	dim3 outerProductBlockDim = dim3(16, 16);
	dim3 outerProductGridDim = dim3( ceil(runningMeanSumLength  / outerProductBlockDim.x),
									 ceil( (runningMeanSumLength / 2) + 1) /  outerProductBlockDim.y);

	for(int i = 0; i < freqChanNum; ++i)
	{
		outerProductKernel<<< gridDim, blockDim >>> (d_outerProduct + (i * covarianceLength), d_runningMeanSum, runningMeanSumLength);
	}

	*/

	return NULL;
}



bool CovarianceMatrixCUDAEngine::hitsContainsZeroes(unsigned int* d_hits, unsigned int hitLength)
{
	int blockDim = 256;
	int gridDim = ceil((float) hitLength / blockDim);

	//Reset d_zeroes to false
	hipMemset(d_zeroes, 0, sizeof(bool));

	checkForZeroesKernel<<< gridDim, blockDim >>>(d_hits, hitLength, d_zeroes);
	hipMemcpy(h_zeroes, d_zeroes, sizeof(bool), hipMemcpyDeviceToHost);

	return h_zeroes;
}



__global__ void outerProductKernel(float* result, float* vec, int vectorLength)
{
	int col = (blockIdx.x * blockDim.x) + threadIdx.x; //column
	int row = (blockIdx.y * blockDim.y) + threadIdx.y; //row

	//check bounds
	if(row >= vectorLength || col >= vectorLength)
		return;

	//transpose
	if(row > col)
	{
		row = vectorLength - row;
		col = row + col;
	}

	//compute the index
	int index = (row * vectorLength + col) - ((row * (row + 1)) / 2);

	//do the outer product calculation and add it too the correct element
	result[index] += vec[row] * vec[col];
}



__global__ void meanStokesKernel(float* d_amps, unsigned int ampsLength, unsigned int* d_hits, unsigned int stokesLength)
{
	int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if(absoluteThreadIdx >= ampsLength)
		return;

	unsigned int hitVal = d_hits[ absoluteThreadIdx / stokesLength ];

	//can't divide by zero so just return
	if(hitVal == 0)
	{
		d_amps[absoluteThreadIdx] = 0;
		return;
	}

	d_amps[absoluteThreadIdx] = d_amps[absoluteThreadIdx] / (float)hitVal;

}



__global__ void applyScaleKernel(float* amps, unsigned int ampsLength, double scaleFactor)
{
	int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if(absoluteThreadIdx >= ampsLength)
		return;

	amps[absoluteThreadIdx] /= scaleFactor;
}



//----PHASE SERIES COMBINE STUFF----


//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, float* original, const float* add)
{
	for(int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}



//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, unsigned int* original, const unsigned int* add)
{
	for(int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}


__global__ void genericDivideKernel(unsigned int n, float* d_numerators, unsigned int denominator)
{
	for(int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		d_numerators[absIdx] /= denominator;
	}
}



__global__ void checkForZeroesKernel(float* d_hits, unsigned int hitsLength, bool* d_zeroes)
{
	for(int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < hitsLength; absIdx += gridDim.x * blockDim.x)
	{
		if(d_hits[absIdx] == 0)
			*d_zeroes = true;
	}
}

