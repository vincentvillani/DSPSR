#include "hip/hip_runtime.h"
/*
 * CovarianceMatrixEngineCUDA.C
 *
 *  Created on: 01/09/2014
 *      Author: vincentvillani
 */

#include "dsp/CovarianceMatrixCUDAEngine.h"

//TODO: VINCENT: ADD A HITS CHAN == 1 VARIATION TO STOP NEEDLESS COPYIES

//#if HAVE_CUDA
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
//#endif


dsp::CovarianceMatrixCUDAEngine::CovarianceMatrixCUDAEngine()
{
	hipMalloc(&d_zeroes, sizeof(bool));
	h_zeroes = false;
}


dsp::CovarianceMatrixCUDAEngine::~CovarianceMatrixCUDAEngine()
{
	hipFree(d_zeroes);
}


//SINGLE HIT DIM
void dsp::CovarianceMatrixCUDAEngine::computeCovarianceMatricesCUDA(const PhaseSeries* ps, CovarianceMatrixResult* cmr)
{
	unsigned int hitsLength = cmr->getHitsLength();
	unsigned int* d_hits = cmr->getHits();
	const unsigned int* h_hits = getHitsPtr(ps, cmr, 0);

	gpuErrchk( hipMemcpy(d_hits, h_hits, sizeof(unsigned int) * hitsLength, hipMemcpyHostToDevice) );

	//If there are bins with zeroes, discard everything
	if ( hitsContainsZeroes(d_hits, hitsLength) )
	{
		printf("There are bins with zeroes, returning...\n");
		return;
	}


	computeCovarianceMatrix(cmr, ps);


	//TODO: VINCENT: DEBUG
	float val;
	hipMemcpy(&val, cmr->getCovarianceMatrix(0), sizeof(float), hipMemcpyDeviceToHost);
	printf("Value: %f\n", val);

	cmr->getPhaseSeries()->combine(ps);

}



void dsp::CovarianceMatrixCUDAEngine::computeCovarianceMatrix(CovarianceMatrixResult* cmr, const PhaseSeries* ps)
{
	unsigned int ampsLength = cmr->getAmpsLength();
	unsigned int covMatrixLength = cmr->getCovarianceMatrixLength();
	unsigned int stokesLength = cmr->getStokesLength();


	float* d_amps = cmr->getAmps();
	unsigned int* d_hits = cmr->getHits();
	float* d_runningMean;
	float* d_result;


	unsigned int meanBlockDim = 256;
	unsigned int meanGridDim =  ceil( ampsLength / meanBlockDim);
	unsigned int outerProductBlockSize = 256;
	unsigned int outerProductGridDim = min( (int)ceil( (int)((ampsLength * (ampsLength + 1)) / 2) / outerProductBlockSize), 65535);


	//compute the covariance matrix for each freq chan
	for(unsigned int i = 0; i < cmr->getNumberOfFreqChans(); ++i)
	{
		//first normalise/compute the mean of the amps by dividing it by the hits
		const float* h_amps = ps->get_datptr(i, 0);
		gpuErrchk(hipMemcpy(d_amps, h_amps + (i * ampsLength), sizeof(float) * ampsLength, hipMemcpyHostToDevice));

		//h_hits values should be copied over to d_hits before this function is called <------------ IS THIS TRUE?????????!!!!!?!?!?!!??
		printf("Launching Mean Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);
		meanStokesKernel <<< meanGridDim, meanBlockDim >>> (d_amps, ampsLength, d_hits, stokesLength);

		//TODO: DEBUG
		hipError_t error = hipPeekAtLastError();
		if(error != hipSuccess)
		{
			printf("CUDA ERROR: %s\n", hipGetErrorString(error));
			exit(1);
		}


		//Add the normalised amps to the running mean
		d_runningMean = cmr->getRunningMeanSum(i);
		genericAddKernel <<< meanGridDim, meanBlockDim >>> (ampsLength, d_runningMean, d_amps);

		//TODO: DEBUG
		error = hipPeekAtLastError();
		if(error != hipSuccess)
		{
			printf("CUDA ERROR: %s\n", hipGetErrorString(error));
			exit(1);
		}



		//Compute the outer product
		printf("Launching outerProduct Kernel with gridDim: %u, blockDim: %u\n\n",
				outerProductGridDim, outerProductBlockSize);
		d_result = cmr->getCovarianceMatrix(i);
		outerProductKernelNew <<<outerProductGridDim, outerProductBlockSize>>>
				(d_result, covMatrixLength, d_amps, ampsLength);

	}

	//combine phase series
	cmr->getPhaseSeries()->combine(ps);
}



float* dsp::CovarianceMatrixCUDAEngine::compute_final_covariance_matrices_device_DEBUG(CovarianceMatrixResult* cmr)
{
	printf("BEFORE PSOP\n");

	float* phaseSeriesOuterProduct = compute_outer_product_phase_series_device_DEBUG(cmr);

	printf("AFTER PSOP\n");

	unsigned int freqChanNum = cmr->getNumberOfFreqChans();
	unsigned int covarianceMatrixLength = cmr->getCovarianceMatrixLength();
	unsigned int unloadCalledNum = cmr->getUnloadCallCount();

	float* covarianceMatrix = new float[covarianceMatrixLength];

	for(int i = 0; i < freqChanNum; ++i)
	{
		float* d_covarianceMatrix = cmr->getCovarianceMatrix(i);
		gpuErrchk(hipMemcpy(covarianceMatrix, d_covarianceMatrix, sizeof(float) * covarianceMatrixLength, hipMemcpyDeviceToHost));

		for(int j = 0; j < covarianceMatrixLength; ++j)
		{

			covarianceMatrix[j] /= unloadCalledNum;
			covarianceMatrix[j] -= phaseSeriesOuterProduct[(i * covarianceMatrixLength) + j];
		}

		//copy results back
		gpuErrchk(hipMemcpy(d_covarianceMatrix, covarianceMatrix, sizeof(float) * covarianceMatrixLength, hipMemcpyHostToDevice));
	}

	printf("HERE\n");



	float* finalCov = new float[covarianceMatrixLength * freqChanNum];
	gpuErrchk(hipMemcpy(finalCov, cmr->getCovarianceMatrix(0), sizeof(float) * covarianceMatrixLength * freqChanNum, hipMemcpyDeviceToHost));

	printf("DONE\n");

	delete[] covarianceMatrix;
	delete[] phaseSeriesOuterProduct;
	return finalCov;

}



float* dsp::CovarianceMatrixCUDAEngine::compute_final_covariance_matrices_device(CovarianceMatrixResult* cmr)
{

	//Compute the phase series outer products
	float* d_phaseSeriesOuterProduct = compute_outer_product_phase_series_device(cmr);

	unsigned int totalElementLength = cmr->getCovarianceMatrixLength() * cmr->getNumberOfFreqChans();
	unsigned int blockDim = 256;
	unsigned int gridDim = min (ceil ( totalElementLength / blockDim), (double) 65535); //number of elements / blockdim

	//Divide all x^2 terms by unload call count
	printf("Launching generic divide kernel with gridDim: %u, blockDim: %u\n", gridDim, blockDim);
	genericDivideKernel <<< gridDim, blockDim >>> (totalElementLength, cmr->getCovarianceMatrix(0), cmr->getUnloadCallCount());

	//TODO: VINCENT: DEBUG
	hipError_t error = hipPeekAtLastError();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(2);
	}


	genericSubtractionKernel <<< gridDim, blockDim >>> (totalElementLength, cmr->getCovarianceMatrix(0), d_phaseSeriesOuterProduct);

	//TODO: VINCENT: DEBUG
	hipError_t error2 = hipPeekAtLastError();
	if(error2 != hipSuccess)
	{
		printf("CUDA ERROR2: %s\n", hipGetErrorString(error2));
		exit(2);
	}

	hipFree(d_phaseSeriesOuterProduct);

	float* h_outerProduct = new float[totalElementLength];
	hipMemcpy(h_outerProduct, cmr->getCovarianceMatrix(0), sizeof(float) * totalElementLength, hipMemcpyDeviceToHost);

	return h_outerProduct;

}




float* dsp::CovarianceMatrixCUDAEngine::compute_outer_product_phase_series_device(CovarianceMatrixResult* cmr)
{

	unsigned int totalCovarianceLength = cmr->getCovarianceMatrixLength() * cmr-> getNumberOfFreqChans();

	float* d_runningMeanSum = cmr->getRunningMeanSum(0);
	unsigned int runningMeanSumLength = cmr->getRunningMeanSumLength();

	float* d_outerProduct;
	hipMalloc(&d_outerProduct, sizeof(float) * totalCovarianceLength);
	hipMemset(d_outerProduct, 0, sizeof(float) * totalCovarianceLength);

	//divide the running mean by the number of times unload was called
	unsigned int blockDim = 256;
	unsigned int gridDim = ceil(runningMeanSumLength / blockDim);

	printf("Starting generic divide kernel - GridDim: %u, BlockDim: %u\n", gridDim, blockDim);
	genericDivideKernel<<< gridDim, blockDim >>> (runningMeanSumLength, d_runningMeanSum, cmr->getUnloadCallCount());

	//TODO: VINCENT: DEBUG
	hipError_t error = hipPeekAtLastError();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(2);
	}

	//Do the outer product

	unsigned int ampsLength = cmr->getAmpsLength();
	unsigned int outerProductBlockDim = 256;
	unsigned int outerProductGridDim = min( (int)ceil( (int)((ampsLength * (ampsLength + 1)) / 2) / outerProductBlockDim), 65535);

	//unsigned int oneFreqRunningMeanLength = cmr->getBinNum() * cmr->getStokesLength();

	for(unsigned int i = 0; i < cmr->getNumberOfFreqChans(); ++i)
	{
		printf("Starting outer product kernel - GridDim: %u, BlockDim: %u\n", outerProductGridDim, outerProductBlockDim);
		outerProductKernelNew <<< outerProductGridDim, outerProductBlockDim >>>
				(d_outerProduct + (i * cmr->getCovarianceMatrixLength()), cmr->getCovarianceMatrixLength(),
						d_runningMeanSum + (i * ampsLength), ampsLength);

		//TODO: VINCENT: DEBUG
		hipError_t error2 = hipPeekAtLastError();
		if(error2 != hipSuccess)
		{
			printf("CUDA ERROR: %s\n", hipGetErrorString(error2));
			exit(2);
		}
	}


	return d_outerProduct;
}




float* dsp::CovarianceMatrixCUDAEngine::compute_outer_product_phase_series_device_DEBUG(CovarianceMatrixResult* cmr)
{
	unsigned int unloadCallCount = cmr->getUnloadCallCount();
	unsigned int freqChanNum = cmr->getNumberOfFreqChans();
	unsigned int covarianceLength = cmr->getCovarianceMatrixLength();
	unsigned int ampsLength = cmr->getBinNum() * cmr->getStokesLength();

	float* outerProduct = new float [freqChanNum * covarianceLength];
	float* runningMeanSum = new float[ampsLength];


	//For each freq channel
	for(unsigned int channel = 0; channel < freqChanNum; ++channel)
	{

		float* d_runningMeanSum = cmr->getRunningMeanSum(channel);
		hipMemcpy(runningMeanSum, d_runningMeanSum, sizeof(float) * ampsLength, hipMemcpyDeviceToHost);


		//divide running mean sum by number of times called
		for(unsigned int i = 0; i < ampsLength; ++i)
		{
			runningMeanSum[i] /= unloadCallCount;
		}


		//Do the outer product
		for(unsigned int row = 0; row < ampsLength; ++row)
		{
			for(unsigned int col = row; col < ampsLength; ++col)
			{
				outerProduct[ (channel * covarianceLength) +  ((row * ampsLength + col) - ( (row * (row + 1)) / 2) ) ] =
						runningMeanSum[row] * runningMeanSum[col];
			}
		}

	}


	return outerProduct;
}




bool dsp::CovarianceMatrixCUDAEngine::hitsContainsZeroes(unsigned int* d_hits, unsigned int hitLength)
{
	int blockDim = 256;
	int gridDim = ceil((float) hitLength / blockDim);

	//Reset d_zeroes to false
	hipMemset(d_zeroes, 0, sizeof(bool));


	checkForZeroesKernel<<< gridDim, blockDim >>> (d_hits, hitLength, d_zeroes);

	//TODO: VINCENT: DEBUG
	hipError_t error2 = hipPeekAtLastError();
	if(error2 != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error2));
		exit(2);
	}


	gpuErrchk(hipMemcpy(&h_zeroes, d_zeroes, sizeof(bool), hipMemcpyDeviceToHost));

	return h_zeroes;
}




const unsigned int* dsp::CovarianceMatrixCUDAEngine::getHitsPtr(const PhaseSeries* phaseSeriesData, CovarianceMatrixResult* covarianceMatrixResult, int freqChan)
{
	//return the only channel
	if(covarianceMatrixResult->getNumberOfHitChans() == 1)
		return phaseSeriesData->get_hits(0);
	else
		return phaseSeriesData->get_hits(freqChan); //Return the hits pointer using the freq channel
}




void dsp::CovarianceMatrixCUDAEngine::outputUpperTriangularMatrix(float* result, unsigned int rowLength, std::string filename)
{

	FILE* file = fopen(filename.c_str(), "w");

	int numZeros = 0;
	int iterator = 0;

	//for every row
	for(int i = 0; i < rowLength; ++i)
	{
		//print preceding zeros
		for(int j = 0; j < numZeros; ++j)
		{
			fprintf(file, "0 ");
		}

		//print array values
		for(int k = 0; k < rowLength - numZeros; ++k)
		{
			fprintf(file, "%f ", result[iterator]);
			++iterator;
		}

		fprintf(file, "\n");
		numZeros++;
	}

	fclose(file);

}




__global__ void outerProductKernel(float* result, float* vec, unsigned int vectorLength)
{
	int col = (blockIdx.x * blockDim.x) + threadIdx.x; //column
	int row = (blockIdx.y * blockDim.y) + threadIdx.y; //row

	//check bounds
	if(row >= vectorLength || col >= vectorLength)
		return;

	//transpose
	if(row > col)
	{
		row = vectorLength - row;
		col = row + col;
	}

	//compute the index
	int index = (row * vectorLength + col) - ((row * (row + 1)) / 2);

	//do the outer product calculation and add it too the correct element
	result[index] += vec[row] * vec[col];
}



__global__ void outerProductKernelNew(float* result, unsigned int resultLength, float* vec, unsigned int vecLength)
{
	for(unsigned int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x; absoluteThreadIdx < resultLength; absoluteThreadIdx += gridDim.x * blockDim.x)
	{
		unsigned int row = absoluteThreadIdx / vecLength;
		unsigned int col = absoluteThreadIdx % vecLength;

		if(row > col)
		{
			row = vecLength - row;
			col = row + col;
		}

		//compute the index
		int index = (row * vecLength + col) - ((row * (row + 1)) / 2);

		//do the outer product calculation and add it too the correct element
		result[index] += vec[row] * vec[col];

	}
}



//(d_amps, ampsLength, d_hits, stokesLength)
__global__ void meanStokesKernel(float* d_amps, unsigned int ampsLength, unsigned int* d_hits, unsigned int stokesLength)
{
	unsigned int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if(absoluteThreadIdx >= ampsLength)
		return;

	unsigned int hitVal = d_hits[ absoluteThreadIdx / stokesLength ];

	d_amps[absoluteThreadIdx] /= hitVal;

}



__global__ void applyScaleKernel(float* amps, unsigned int ampsLength, double scaleFactor)
{
	unsigned int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

	if(absoluteThreadIdx >= ampsLength)
		return;

	amps[absoluteThreadIdx] /= scaleFactor;
}



//----PHASE SERIES COMBINE STUFF----


//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, float* original, const float* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}



//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, unsigned int* original, const unsigned int* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}



__global__ void genericSubtractionKernel(unsigned int n, float* original, const float* sub)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] -= sub[absIdx];
	}
}


__global__ void genericDivideKernel(unsigned int n, float* d_numerators, unsigned int denominator)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		d_numerators[absIdx] /= denominator;
	}
}



__global__ void checkForZeroesKernel(unsigned int* d_hits, unsigned int hitsLength, bool* d_zeroes)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < hitsLength; absIdx += gridDim.x * blockDim.x)
	{
		if(d_hits[absIdx] == 0)
		{
			//printf("ZERO KERNEL VAL: %u\n", d_hits[absIdx]);
			*d_zeroes = true;
		}
	}
}

