#include "hip/hip_runtime.h"
/*
 * TimeSeriesCombinerCUDA.C
 *
 *  Created on: 21/09/2014
 *      Author: vincentvillani
 */

#include "dsp/TimeSeriesCombinerCUDA.h"

dsp::TimeSeriesCombinerCUDA::TimeSeriesCombinerCUDA()
{

}


dsp::TimeSeriesCombinerCUDA::~TimeSeriesCombinerCUDA()
{

}


void dsp::TimeSeriesCombinerCUDA::combine(TimeSeries* lhs, const TimeSeries* rhs)
{
	if(lhs == NULL || rhs == NULL)
		return;

	//TODO: ASK WILLEM ABOUT THIS
	/*
	if(lhs->get_ndat() == 0)
	{
		lhs->operator=(rhs);
		return;
	}
	*/

	if(!lhs->combinable(*lhs))
	{
		return;
	}

	if(lhs->get_ndat() != rhs->get_ndat())
	{
		return;
	}

	uint64_t npt = lhs->get_ndat() * lhs->get_ndim();
	unsigned int blockDim = 256;
	unsigned int gridDim;

	float* d_data1;


	if(lhs->get_order() == dsp::TimeSeries::OrderTFP)
	{
		npt *= lhs->get_nchan() * lhs->get_npol();
		gridDim = min ( (unsigned int)ceil(npt / blockDim), 65535);

		d_data1 = lhs->get_dattfp();
		const float* d_data2 = rhs->get_dattfp();

		printf("Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
		genericAddKernel <<< gridDim, blockDim >>> (npt, d_data1, d_data2);

		//TODO: VINCENT: DEBUG
		hipError_t error2 = hipPeekAtLastError();
		if(error2 != hipSuccess)
		{
			printf("CUDA ERROR: %s\n", hipGetErrorString(error2));
			exit(2);
		}

		return;
	}


	gridDim = min ( (unsigned int)ceil(npt / blockDim), 65535);

	for (unsigned ichan = 0; ichan < lhs->get_nchan(); ++ichan)
	{
		for (unsigned ipol = 0; ipol < lhs->get_npol(); ++ipol)
		{
			d_data1 = lhs->get_datptr (ichan, ipol);
			const float* d_data2 = rhs->get_datptr (ichan, ipol);

			printf("TIME SERIES COMBINE: Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
			genericAddKernel <<< gridDim, blockDim >>> (npt, d_data1, d_data2);

			//TODO: VINCENT: DEBUG
			hipError_t error2 = hipPeekAtLastError();
			if(error2 != hipSuccess)
			{
				printf("CUDA ERROR: %s\n", hipGetErrorString(error2));
				exit(2);
			}
		}
	}
}




/*
//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, float* original, const float* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}


//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, unsigned int* original, const unsigned int* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}
*/


