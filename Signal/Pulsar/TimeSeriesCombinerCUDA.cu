#include "hip/hip_runtime.h"
/*
 * TimeSeriesCombinerCUDA.C
 *
 *  Created on: 21/09/2014
 *      Author: vincentvillani
 */

#include "dsp/TimeSeriesCombinerCUDA.h"

dsp::TimeSeriesCombinerCUDA::TimeSeriesCombinerCUDA()
{

}


dsp::TimeSeriesCombinerCUDA::~TimeSeriesCombinerCUDA()
{

}


void dsp::TimeSeriesCombinerCUDA::combine(TimeSeries* lhs, const TimeSeries* rhs)
{
	if(lhs == NULL || rhs == NULL)
		return;

	/*
	if(lhs->get_ndat() == 0)
	{
		lhs->operator=(rhs);
		return;
	}
	*/

	if(!lhs->combinable(*lhs))
	{
		return;
	}

	if(lhs->get_ndat() != rhs->get_ndat())
	{
		return;
	}

	uint64_t npt = lhs->get_ndat() * lhs->get_ndim();
	unsigned int blockDim = 256;
	unsigned int gridDim;

	if(lhs->get_order() == dsp::TimeSeries::OrderTFP)
	{
		npt *= lhs->get_nchan() * lhs->get_npol();
		gridDim = min ( (unsigned int)ceil(npt / blockDim), 65535);


		//TODO: VINCENT, THIS WILL ALREADY BE ON THE DEVICE IN THE FINAL VERSION, NO NEED FOR COPIES
		float* h_data1 = lhs->get_dattfp();
		float* h_data2 = rhs->get_dattfp();
		float* d_data1;
		float* d_data2;

		hipMalloc(&d_data1, sizeof(float) * npt);
		hipMalloc(&d_data2, sizeof(float) * npt);

		hipMemcpy(d_data1, h_data1, sizeof(float) * npt, hipMemcpyHostToDevice);
		hipMemcpy(d_data2, h_data2, sizeof(float) * npt, hipMemcpyHostToDevice);

		printf("Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
		genericAddKernel <<< gridDim, blockDim >>> (npt, d_data1, d_data2);

		hipMemcpy(h_data1, d_data1, sizeof(float) * npt, hipMemcpyDeviceToHost);

		hipFree(d_data1);
		hipFree(d_data2);

		return;
	}


	float* h_data1;
	float* h_data2;
	float* d_data1;
	float* d_data2;

	hipMalloc(&d_data1, sizeof(float) * npt);
	hipMalloc(&d_data2, sizeof(float) * npt);

	gridDim = min ( (unsigned int)ceil(npt / blockDim), 65535);

	for (unsigned ichan = 0; ichan < lhs->get_nchan(); ichan++)
	{
		for (unsigned ipol = 0; ipol < lhs->get_npol(); ipol++)
		{
			//TODO: VINCENT, THIS WILL ALREADY BE ON THE DEVICE IN THE FINAL VERSION, NO NEED FOR COPIES
			h_data1 = lhs->get_datptr (ichan, ipol);
			h_data2 = rhs->get_datptr (ichan, ipol);

			hipMemcpy(d_data1, h_data1, sizeof(float) * npt, hipMemcpyHostToDevice);
			hipMemcpy(d_data2, h_data2, sizeof(float) * npt, hipMemcpyHostToDevice);

			printf("Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
			genericAddKernel <<< gridDim, blockDim >>> (npt, d_data1, d_data2);

			hipMemcpy(h_data1, d_data1, sizeof(float) * npt, hipMemcpyDeviceToHost);
		}
	}

	hipFree(d_data1);
	hipFree(d_data2);
}





//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, float* original, const float* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}


//Kernel for generically adding things on the GPU
__global__ void genericAddKernel(unsigned int n, unsigned int* original, const unsigned int* add)
{
	for(unsigned int absIdx = blockDim.x * blockIdx.x + threadIdx.x; absIdx < n; absIdx += gridDim.x * blockDim.x)
	{
		original[absIdx] += add[absIdx];
	}
}


