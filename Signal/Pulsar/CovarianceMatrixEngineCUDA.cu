#include "hip/hip_runtime.h"
/*
 * CovarianceMatrixEngineCUDA.C
 *
 *  Created on: 01/09/2014
 *      Author: vincentvillani
 */



#include "dsp/CovarianceMatrixEngineCUDA.h"

void computeCovarianceMatrixCUDAEngine(float* d_resultVector, unsigned int resultElementOffset,
		const float* h_amps, float* d_amps, unsigned int ampsLength,
		 const unsigned int* h_hits, unsigned int* d_hits, unsigned int hitsLength,
		 unsigned int stokesLength, double scaleFactor, unsigned int blockDim2D)
{

	//printf("RUNNING KERNELS\n");

	int meanBlockDim = 256;
	int meanGridDim = ceil((float) ampsLength / meanBlockDim);

	//Copy data to device
	hipMemcpy(d_amps, h_amps, sizeof(float) * ampsLength, hipMemcpyHostToDevice);
	hipMemcpy(d_hits, h_hits, sizeof(unsigned int) * hitsLength, hipMemcpyHostToDevice);

	printf("Launching scale Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);
	//applyScale <<< meanGridDim, meanBlockDim >>> (d_amps, ampsLength, scaleFactor);


	printf("Launching Mean Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);
	meanStokesKernel<<< meanGridDim, meanBlockDim >>>(d_amps, ampsLength, d_hits, stokesLength);

	//TODO: DEBUG
	hipError_t error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(1);
	}

	//Compute the needed block and grid dimensions
	int blockDimX = blockDim2D;
	int blockDimY = blockDim2D;
	int gridDimX = ceil((float) ampsLength / blockDimX);
	int gridDimY = ceil((float) ((ampsLength / 2) + 1) / blockDimY);

	dim3 grid = dim3(gridDimX, gridDimY);
	dim3 block = dim3(blockDimX, blockDimY);

	//Call the kernel
	//Compute covariance matrix
	printf("Launching outerProduct Kernel with gridDim: (%d, %d), blockDim: (%d, %d)\n\n",
			grid.x, grid.y, block.x, block.y);
	outerProductKernel<<< grid, block >>>(d_resultVector + resultElementOffset, d_amps, ampsLength);

	//TODO: DEBUG
	error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
		exit(1);
	}

}
