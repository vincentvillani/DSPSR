#include "hip/hip_runtime.h"
/*
 * CovarianceMatrixEngineCUDA.C
 *
 *  Created on: 01/09/2014
 *      Author: vincentvillani
 */



#include "dsp/CovarianceMatrixEngineCUDA.h"

void computeCovarianceMatrixCUDAEngine(float* d_resultVector, unsigned int resultElementOffset,
		const float* h_amps, float* d_amps, unsigned int ampsLength,
		 const unsigned int* h_hits, unsigned int* d_hits, unsigned int hitsLength,
		 unsigned int stokesLength, double scaleFactor, unsigned int blockDim2D)
{

	printf("RUNNING KERNELS\n");

	int meanBlockDim = 256;
	int meanGridDim = ceil((float) ampsLength / meanBlockDim);

	//Copy data to device
	hipMemcpy(d_amps, h_amps, sizeof(float) * ampsLength, hipMemcpyHostToDevice);
	hipMemcpy(d_hits, h_hits, sizeof(unsigned int) * hitsLength, hipMemcpyHostToDevice);

	//unsigned int hitZero;
	//hipMemcpy(&hitZero, d_hits, sizeof(unsigned int), hipMemcpyDeviceToHost);

	//printf("Hit Zero: %d\n", hitZero);

	//printf("Launching scale Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);

	applyScale <<< meanGridDim, meanBlockDim >>> (d_amps, ampsLength, scaleFactor);


	//printf("Launching Mean Kernel with gridDim: %d, blockDim: %d\n", meanGridDim, meanBlockDim);

	//float ampZero;

	//hipMemcpy(&ampZero, d_amps, sizeof(float), hipMemcpyDeviceToHost);

	//printf("After: amp zero: %f\n", ampZero);
	//printf("After: hit zero: %d\n", h_hits[0]);


	meanStokesKernel<<< meanGridDim, meanBlockDim >>>(d_amps, ampsLength, d_hits, stokesLength);

	//TODO: DEBUG
	hipError_t error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
	}

	//Compute the needed block and grid dimensions
	int blockDimX = blockDim2D;
	int blockDimY = blockDim2D;
	int gridDimX = ceil((float) ampsLength / blockDimX);
	int gridDimY = ceil((float) ((ampsLength / 2) + 1) / blockDimY);

	dim3 grid = dim3(gridDimX, gridDimY);
	dim3 block = dim3(blockDimX, blockDimY);

	printf("Launching outerProduct Kernel with gridDim: (%d, %d), blockDim: (%d, %d)\n",
			grid.x, grid.y, block.x, block.y);

	//Call the kernel
	//Compute covariance matrix
	outerProductKernel<<< grid, block >>>(d_resultVector + resultByteOffset, d_amps, ampsLength);

	//TODO: DEBUG
	error = hipDeviceSynchronize();
	if(error != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
	}

}
