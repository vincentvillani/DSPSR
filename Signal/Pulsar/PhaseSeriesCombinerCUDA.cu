#include "hip/hip_runtime.h"
/*
 * PhaseSeriesCombinerCUDA.C
 *
 *  Created on: 21/09/2014
 *      Author: vincentvillani
 */

#include "dsp/PhaseSeriesCombinerCUDA.h"
#include "dsp/PhaseSeries.h"
#include "dsp/CovarianceMatrixKernels.h"
#include "dsp/Memory.h"


dsp::PhaseSeriesCombinerCUDA::PhaseSeriesCombinerCUDA()
{
	_tsc = new dsp::TimeSeriesCombinerCUDA();
	d_temp_data1 = NULL;
	d_temp_data2 = NULL;

}



dsp::PhaseSeriesCombinerCUDA::~PhaseSeriesCombinerCUDA()
{
	delete _tsc;

	if(d_temp_data1 != NULL)
		hipFree(d_temp_data1);

	if(d_temp_data2 != NULL)
		hipFree(d_temp_data2);
}



void dsp::PhaseSeriesCombinerCUDA::combine(PhaseSeries* const lhs, const PhaseSeries* rhs)
{
	if(lhs == NULL || rhs == NULL)
	{
		//printf("Returning 1\n");
		return;
	}

	if(rhs->get_nbin() == 0 || rhs->get_integration_length() == 0.0)
	{
		//printf("Returning 2\n");
		return;
	}

	if( !lhs->mixable(*rhs, rhs->get_nbin() ) )
	{
		//printf("Returning 3\n");
		return;
	}


	//TODO: VINCENT: ADD THIS PART BACK IN
	//combine the time series part
	_tsc->combine(lhs, rhs);

	const unsigned int hitLength = lhs->get_nbin() * lhs->hits_nchan;
	unsigned int nHitChan = lhs->get_hits_nchan();
	unsigned int totalHitLength = hitLength * nHitChan;

	unsigned int* h_lhsHits = lhs->hits;
	unsigned int* h_rhsHits = rhs->hits;

	unsigned int blockDim = 256;
	unsigned int gridDim = min ( (unsigned int)ceil(totalHitLength / blockDim), 65535);



	if(rhs->get_memory()->on_host())
		printf("RHS MEMORY IS ON HOST\n");
	else
		printf("RHS MEMORY IS NOT ON HOST\n");


	//TODO: VINCENT: NO NEED TO DO THIS IN THE FINAL VERSION
	if(d_temp_data1 == NULL || d_temp_data2 == NULL)
	{
		hipMalloc(&d_temp_data1, sizeof(unsigned int) * totalHitLength);
		hipMalloc(&d_temp_data2, sizeof(unsigned int) * totalHitLength);
	}

	//TODO: VINCENT: NO NEED TO DO THIS IN THE FINAL VERSION
	hipMemcpy(d_temp_data1, h_lhsHits, sizeof(unsigned int) * totalHitLength, hipMemcpyHostToDevice);
	hipMemcpy(d_temp_data2, h_rhsHits, sizeof(unsigned int) * totalHitLength, hipMemcpyHostToDevice);

	printf("PHASE SERIES COMBINE: Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
	genericAddKernel <<<gridDim, blockDim>>> (totalHitLength, d_temp_data1, d_temp_data2);

	//TODO: VINCENT: DEBUG
	hipError_t error2 = hipPeekAtLastError();
	if(error2 != hipSuccess)
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error2));
		exit(2);
	}

	//TODO: VINCENT: NO NEED TO DO THIS IN THE FINAL VERSION
	//copy the data back to the host
	hipMemcpy(h_lhsHits, d_temp_data1, sizeof(unsigned int) * totalHitLength, hipMemcpyDeviceToHost);

	/*
	for(unsigned int i = 0; i < nHitChan; ++i)
	{
		/*
		if(lhs->get_memory()->on_host())
		{
			printf("MEMORY IS ON HOST\n");
		}
		else
			printf("MEMORY IS NOT ON HOST\n");


		//TODO: VINCENT NO NEED TO COPY IN THE FINAL VERSION?

		printf("PHASE SERIES COMBINE: Launching GenericAddKernel with Grid Dim: %u, Block Dim: %u\n", gridDim, blockDim);
		genericAddKernel <<<gridDim, blockDim>>> (hitLength, d_lhsHits + (i * hitLength), d_rhsHits + (i * hitLength));
	}
*/

	lhs->integration_length += rhs->integration_length;
	lhs->ndat_total += rhs->ndat_total;

	if (!lhs->ndat_expected)
		lhs->ndat_expected = rhs->ndat_expected;

}







